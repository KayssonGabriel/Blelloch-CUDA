#include <iostream>
#include <vector>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>

__global__ void blelloch_up_sweep(int* array, int tamanho, int* trabalho, int* passos)
{
    int passo = 1;
    while (passo < tamanho)
    {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        int offset = 2 * passo * index;
        if (offset + 2 * passo - 1 < tamanho)
        {
            array[offset + 2 * passo - 1] += array[offset + passo - 1];
            atomicAdd(trabalho, 1); // Contabiliza o trabalho da soma
        }
        if (index == 0) atomicAdd(passos, 1); // Contabiliza um passo por iteração
        passo *= 2;
        __syncthreads();
    }
}

__global__ void blelloch_down_sweep(int* array, int tamanho, int* trabalho, int* passos)
{
    array[tamanho - 1] = 0;
    int passo = tamanho / 2;
    while (passo > 0)
    {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        int offset = 2 * passo * index;
        if (offset + 2 * passo - 1 < tamanho)
        {
            int temp = array[offset + passo - 1];
            array[offset + passo - 1] = array[offset + 2 * passo - 1];
            array[offset + 2 * passo - 1] += temp;
            atomicAdd(trabalho, 1); // Contabiliza o trabalho da troca
        }
        if (index == 0) atomicAdd(passos, 1); // Contabiliza um passo por iteração
        passo /= 2;
        __syncthreads();
    }
}

void medir_tempo_blelloch(int tamanho)
{
    int* h_array = new int[tamanho];
    std::fill_n(h_array, tamanho, 1);

    int* d_array;
    hipMalloc((void**)&d_array, tamanho * sizeof(int));
    hipMemcpy(d_array, h_array, tamanho * sizeof(int), hipMemcpyHostToDevice);

    int h_trabalho = 0, h_passos = 0;
    int *d_trabalho, *d_passos;
    hipMalloc((void**)&d_trabalho, sizeof(int));
    hipMalloc((void**)&d_passos, sizeof(int));
    hipMemcpy(d_trabalho, &h_trabalho, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_passos, &h_passos, sizeof(int), hipMemcpyHostToDevice);

    auto inicio = std::chrono::high_resolution_clock::now();
    int blocos = (tamanho + 255) / 256;
    blelloch_up_sweep<<<blocos, 256>>>(d_array, tamanho, d_trabalho, d_passos);
    hipDeviceSynchronize();
    blelloch_down_sweep<<<blocos, 256>>>(d_array, tamanho, d_trabalho, d_passos);
    hipDeviceSynchronize();
    auto fim = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> duracao = fim - inicio;

    hipMemcpy(&h_trabalho, d_trabalho, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_passos, d_passos, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << std::fixed << std::setprecision(6);
    std::cout << "Tamanho do array: " << tamanho
        << " | Tempo (Blelloch): " << duracao.count() << " segundos"
        << " | Trabalho: " << h_trabalho
        << " | Passos: " << h_passos << std::endl;

    hipFree(d_array);
    hipFree(d_trabalho);
    hipFree(d_passos);
    delete[] h_array;
}

int main()
{
    int tamanhos[] = {100, 1000, 10000, 100000, 1000000, 10000000};
    for (int tamanho : tamanhos)
    {
        medir_tempo_blelloch(tamanho);
    }
    return 0;
}
